#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <chrono>
#define NUM_DATA 1024

__global__ void vecAdd(int* _a, int* _b, int* _c) {
    int tID = threadIdx.x;
    _c[tID] = _a[tID] + _b[tID];
}

void vecAdd_cpu(int* _a, int* _b, int* _c) {
    for (int i = 0; i < NUM_DATA; i++) {
        _c[i] = _a[i] + _b[i];
    }
}

int main(void) {
    int* a, * b, * c, * hc; // Vectors on the host
    int* da, * db, * dc; // Vectors on the device

    int memSize = sizeof(int) * NUM_DATA;
    printf("%d elements, memSize = %d bytes\n", NUM_DATA, memSize);

    // Memory allocation on the host side
    a = new int[NUM_DATA]; memset(a, 0, memSize);
    b = new int[NUM_DATA]; memset(b, 0, memSize);
    c = new int[NUM_DATA]; memset(c, 0, memSize);
    hc = new int[NUM_DATA]; memset(hc, 0, memSize);

    for (int i = 0; i < NUM_DATA; i++) {
        a[i] = rand() % 10;
        b[i] = rand() % 10;
    }

    auto start_cpu = std::chrono::high_resolution_clock::now();
    vecAdd_cpu(a, b, hc);
    auto stop_cpu = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double, std::milli> cpu_time = stop_cpu - start_cpu;
    printf("CPU execution time: %.6f ms\n", cpu_time.count());

    // Memory allocation on the device side
    hipMalloc(&da, memSize); hipMemset(da, 0 ,memSize);
    hipMalloc(&db, memSize); hipMemset(db, 0, memSize);
    hipMalloc(&dc, memSize); hipMemset(dc, 0, memSize);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);

    // Copy data from host to device
    hipMemcpy(da, a, memSize, hipMemcpyHostToDevice);
    hipMemcpy(db, b, memSize, hipMemcpyHostToDevice);

    // Kernel invocation
    vecAdd<<<1, NUM_DATA>>>(da, db, dc);

    // Copy data from device to host
    hipMemcpy(c, dc, memSize, hipMemcpyDeviceToHost);

    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float gpu_time;
    hipEventElapsedTime(&gpu_time, start, stop);
    printf("Total GPU processing time (with memory transfers): %.6f ms\n", gpu_time);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    // Release device memory
    hipFree(da); hipFree(db); hipFree(dc);

    // Check the result
    bool result = true;
    for (int i = 0; i < NUM_DATA; i++) {
        if (hc[i] != c[i]) {
            result = false;
            printf("[%d] The result is not matched!: host = %d, device = %d\n", i, hc[i], c[i]);
        }
    }

    if (result) {
        printf("GPU calculation is correct!\n");
    }

    printf("\n==== Performance Comparison ====\n");
    printf("CPU time: %.6f ms\n", cpu_time.count());
    printf("GPU time (with mem transfers): %.6f ms\n", gpu_time);
    if (cpu_time.count() > gpu_time)
        printf("=> GPU is faster by %.6f ms\n", cpu_time.count() - gpu_time);
    else
        printf("=> CPU is faster by %.6f ms\n", gpu_time - cpu_time.count());
    
    /* 
    CPU: AMD Ryzen 7 7800X3D
    GPU: GeForce RTX 4070Ti Super
    
    Sample Output:
    1024 elements, memSize = 4096 bytes
    CPU execution time: 0.001779 ms
    Total GPU processing time (with memory transfers): 1.145088 ms
    GPU calculation is correct!

    ==== Performance Comparison ====
    CPU time: 0.001779 ms
    GPU time (with mem transfers): 1.145088 ms
    => CPU is faster by 1.143309 ms
    */
    

    // Release host memory
    delete[] a; delete[] b; delete[] c; delete[] hc;

    return 0;
}
