#include <hip/hip_runtime.h>

#include <stdio.h>

__global__ void printData(int* _dDataPtr) {
    printf("%d ", _dDataPtr[threadIdx.x]);
}

__global__ void setData(int* _dDataPtr) {
    _dDataPtr[threadIdx.x] = 2;
}

int main(void) {
    int data[10] = { 0 };
    for (int i = 0; i < 10; i++) data[i] = 1;

    int* dDataPtr;
    hipMalloc(&dDataPtr, sizeof(int) * 10);
    hipMemset(dDataPtr, 0, sizeof(int) * 10);

    printf("Data in device: ");
    printData<<<1, 10>>>(dDataPtr); // expect: 0 0 0 0 0 0 0 0 0 0

    hipMemcpy(dDataPtr, data, sizeof(int) * 10, hipMemcpyHostToDevice);
    printf("\nHost -> Device: ");
    printData<<<1, 10>>>(dDataPtr); // expect: 1 1 1 1 1 1 1 1 1 1

    setData<<<1, 10>>>(dDataPtr);

    hipMemcpy(data, dDataPtr, sizeof(int) * 10, hipMemcpyDeviceToHost);
    printf("\nDevice -> Host: ");
    for (int i = 0; i < 10; i++) printf("%d ", data[i]); // expect: 2 2 2 2 2 2 2 2 2 2

    printf("\n");

    hipFree(dDataPtr);

    return 0;
}