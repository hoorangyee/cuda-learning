#include "hip/hip_runtime.h"

#include <stdio.h>

void checkDeviceMemory(void){
    size_t free, total;

    hipMemGetInfo(&free, &total);
    printf("Device memory (free/total) = %ld/%ld bytes\n", free, total);
}

int main(void){
    int *dDataPtr;
    hipError_t errorCode;

    checkDeviceMemory();
    errorCode = hipMalloc(&dDataPtr, sizeof(int) * 1024 * 1024);
    printf("hipMalloc - %s\n", hipGetErrorString(errorCode));
    checkDeviceMemory();

    errorCode = hipMemset(dDataPtr, 0, sizeof(int) * 1024 * 1024);
    printf("cudaMemSet - %s\n", hipGetErrorString(errorCode));

    errorCode = hipFree(dDataPtr);
    printf("hipFree - %s\n", hipGetErrorString(errorCode));
    checkDeviceMemory();

    return 0;
}