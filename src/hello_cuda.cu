#include "hip/hip_runtime.h"

#include <stdio.h>

__global__ void helloCUDA() {
    printf("Hello CUDA from GPU!\n");
}

int main() {
    printf("Hello CUDA from CPU!\n");

    helloCUDA<<<2, 5>>>();

    hipError_t e = hipGetLastError();
    e = hipDeviceSynchronize();

    return 0;
}
